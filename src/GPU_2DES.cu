#include "HEOM_utilize.h"
#include "HEOM_kernel_func.h"
#include "HEOM_constant.h"
#include "HEOM_dynamics.h"
#include "HEOM_TD_hamiltonian.h"
#include "HEOM_param.h"
#include "HEOM_polar.h"
#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <unordered_map>
#include <algorithm>


using namespace std;

int main(int argc, char** argv)
{
	string filename(argv[1]);
	param k;
	k.param_2des(filename);
	construct_ADO_set(k);

	hipblasHandle_t cublasH;
	hipblasCreate(&cublasH);

	vector<vector<data_type>> p;
	for (int i = 0; i < k.n_sample; i++) {
		//cout << "Now running sample " << i + 1 << '\n';
		//cout << "The program is running with " << k.ado.size() << " ADOs" << '\n';
		//cout << "The coherent time is " << k.pulses[1].tau0 - k.pulses[0].tau0 << " fs " << "and the population time is ";
		//if (k.pulses[1].tau0 >= k.pulses[0].tau0) cout << k.pulses[2].tau0 - k.pulses[1].tau0 << " fs" << '\n';
		//else cout << k.pulses[2].tau0 - k.pulses[0].tau0 << " fs" << '\n';

		vector<data_type> H(k.sys_size * k.sys_size);
		construct_Hal(k, H);
		//cout << "The disordered Hamiltonain is:" << '\n';
		//print_Hal(H, sys_size);
		//cout << '\n';
		data_type* d_H = To_Device(H);
		polar_mat_set(k);
		polar_mat_ranrot(k);


		vector<data_type> p1;
		twoD_spectrum_solver(k, d_H, 1, 1, 1, p1);

		vector<data_type> p2;
		twoD_spectrum_solver(k, d_H, 1, 1, 0, p2);

		vector<data_type> p3;
		twoD_spectrum_solver(k, d_H, 1, 0, 1, p3);

		vector<data_type> p_i;
		for (int j = 0; j < p1.size(); j++) {
			float real = p1[j].x - p2[j].x - p3[j].x;
			float imag = p1[j].y - p2[j].y - p3[j].y;
			p_i.push_back(make_hipComplex(real, imag));
		}
		p.push_back(p_i);

		hipError_t(hipFree(d_H), __FILE__, __LINE__);
	}

	vector<data_type> P(p[0].size(), make_hipComplex(0.0, 0.0));
	for (int i = 0; i < p.size(); i++) {
		for (int j = 0; j < p[i].size(); j++) {
			P[j].x += p[i][j].x / k.n_sample;
			P[j].y += p[i][j].y / k.n_sample;
		}
	}

	for (int i = 0; i < P.size(); i++) cout << P[i].x << " " << P[i].y << '\n';

	cublasError(hipblasDestroy(cublasH), __FILE__, __LINE__);
	hipError_t(hipDeviceSynchronize(), __FILE__, __LINE__);

	k.param_free();
	return 0;
}



